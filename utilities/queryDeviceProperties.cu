#include <stdio.h> 
#include <iostream>




int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

  }
}

/**

See [hipDeviceProp_t Struct Reference](https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html)

int  ECCEnabled
int  accessPolicyMaxWindowSize
int  asyncEngineCount
int  canMapHostMemory
int  canUseHostPointerForRegisteredMem
int  clockRate
int  computeMode
int  computePreemptionSupported
int  concurrentKernels
int  concurrentManagedAccess
int  cooperativeLaunch
int  cooperativeMultiDeviceLaunch
int  deviceOverlap
int  directManagedMemAccessFromHost
int  globalL1CacheSupported
int  hostNativeAtomicSupported
int  integrated
int  isMultiGpuBoard
int  kernelExecTimeoutEnabled
int  l2CacheSize
int  localL1CacheSupported
char  luid[8]
unsigned int  luidDeviceNodeMask
int  major
int  managedMemory
int  maxBlocksPerMultiProcessor
int  maxGridSize[3]
int  maxSurface1D
int  maxSurface1DLayered[2]
int  maxSurface2D[2]
int  maxSurface2DLayered[3]
int  maxSurface3D[3]
int  maxSurfaceCubemap
int  maxSurfaceCubemapLayered[2]
int  maxTexture1D
int  maxTexture1DLayered[2]
int  maxTexture1DLinear
int  maxTexture1DMipmap
int  maxTexture2D[2]
int  maxTexture2DGather[2]
int  maxTexture2DLayered[3]
int  maxTexture2DLinear[3]
int  maxTexture2DMipmap[2]
int  maxTexture3D[3]
int  maxTexture3DAlt[3]
int  maxTextureCubemap
int  maxTextureCubemapLayered[2]
int  maxThreadsDim[3]
int  maxThreadsPerBlock
int  maxThreadsPerMultiProcessor
size_t  memPitch
int  memoryBusWidth
int  memoryClockRate
int  minor
int  multiGpuBoardGroupID
int  multiProcessorCount
char  name[256]
int  pageableMemoryAccess
int  pageableMemoryAccessUsesHostPageTables
int  pciBusID
int  pciDeviceID
int  pciDomainID
int  persistingL2CacheMaxSize
int  regsPerBlock
int  regsPerMultiprocessor
size_t  reservedSharedMemPerBlock
size_t  sharedMemPerBlock
size_t  sharedMemPerBlockOptin
size_t  sharedMemPerMultiprocessor
int  singleToDoublePrecisionPerfRatio
int  streamPrioritiesSupported
size_t  surfaceAlignment
int  tccDriver
size_t  textureAlignment
size_t  texturePitchAlignment
size_t  totalConstMem
size_t  totalGlobalMem
int  unifiedAddressing
hipUUID  uuid
int  warpSize
*/

